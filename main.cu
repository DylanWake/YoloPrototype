#include <iostream>
#include <torch/torch.h>

using namespace torch;

int main() {
    Tensor test = torch::randn({2, 3}).to(torch::device(torch::Device("cuda")));
    float testDa[] = {1,2,3,4,5,6};
    hipMemcpy(test.data_ptr(), testDa, sizeof(float) * 6, hipMemcpyHostToDevice);
    std::cout << test << std::endl;
    return 0;
}
